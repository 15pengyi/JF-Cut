#include "hip/hip_runtime.h"

/***********************************************************************************************
 * * Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       **
 * * paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **
 * *                                                                                          **
 * * Copyright (c) 2008 International Institute of Information Technology.                    **
 * * All rights reserved.                                                                     **
 * *                                                                                          **
 * * Permission to use, copy, modify and distribute this software and its documentation for   **
 * * educational purpose is hereby granted without fee, provided that the above copyright     **
 * * notice and this permission notice appear in all copies of this software and that you do  **
 * * not sell the software.                                                                   **
 * *                                                                                          **
 * * THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
 * * OTHERWISE.                                                                               **
 * *                                                                                          **
 * * Created By Vibhav Vineet.                                                                **
 * ********************************************************************************************/

#ifndef _PUSHRELABEL_KERNEL_CU_
#define _PUSHRELABEL_KERNEL_CU_

#include "CudaCuts.h"

/************************************************
 * Push operation is performed                 ** 
 * *********************************************/

__global__ void
kernel_push1_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, 
		int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1 )
{
		int x1 = threadIdx.x ;
		int y1 = threadIdx.y ;
		int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		int thid = __umul24( y , width1 ) + x ;

		__shared__ int height_fn[356];

		
		int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

		height_fn[temp_mult] = g_graph_height[thid] ;

		(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
		(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
		(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
		(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

		__syncthreads();

		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[thid] ;


		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}else atomicSub(&g_pull_left[thid-1], 1) ;

			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}else atomicSub( &g_pull_down[thid+width1], 1) ;
		}

}

__global__ void
kernel_relabel_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, 
		int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1 )
{
	int x1 = threadIdx.x ;
	int y1 = threadIdx.y ;
	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;

	__shared__ int height_fn[356];


	int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

	height_fn[temp_mult] = g_graph_height[thid] ;

	(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
	(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
	(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
	(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

	__syncthreads();


	int min_flow_pushed = g_left_weight[thid] ;
	int flow_push = g_push_reser[thid] ;

	if(flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
		g_relabel_mask[thid] = 2 ;
	else
	{
		( flow_push > 0 && ( ( (height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && g_left_weight[thid] > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;
	}


	__syncthreads();

	if(thid < graph_size1 && x < width - 1  && x > 0 && y < rows - 1  && y > 0  )
	{
		if(g_sink_weight[thid] > 0)
		{
			g_height_write[thid] = 1 ;
		}
		else
		{
			int min_height = graph_size ;
			(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1] ) ? min_height = height_fn[temp_mult - 1] : 0 ;
			(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0 ;
			(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34] ) ? min_height = height_fn[temp_mult + 34] : 0 ;
			(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34] ) ? min_height = height_fn[temp_mult - 34] : 0 ;
			g_height_write[thid] = min_height + 1 ;
		}
	}

}


__global__ void
kernel_relabel_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, /*int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, */
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_stochastic, int *g_block_num )
{
		int x1 = threadIdx.x ;
		int y1 = threadIdx.y ;
		int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		int thid = __umul24( y , width1 ) + x ;

		__shared__ int height_fn[356];


		int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

		height_fn[temp_mult] = g_graph_height[thid] ;

		(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
		(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
		(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
		(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

		__syncthreads();


		int min_flow_pushed = g_left_weight[thid] ;
		int flow_push = g_push_reser[thid] ;

		if(flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
			g_relabel_mask[thid] = 2 ;
		else
		{
			( flow_push > 0 && ( ( (height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && g_left_weight[thid] > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;
		}


		__syncthreads();

		if(thid < graph_size1 && x < width - 1  && x > 0 && y < rows - 1  && y > 0  )
		{
			if(g_sink_weight[thid] > 0)
			{
				g_height_write[thid] = 1 ;
			}
			else
			{
				int min_height = graph_size ;
				(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1] ) ? min_height = height_fn[temp_mult - 1] : 0 ;
				(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0 ;
				(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34] ) ? min_height = height_fn[temp_mult + 34] : 0 ;
				(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34] ) ? min_height = height_fn[temp_mult - 34] : 0 ;
				g_height_write[thid] = min_height + 1 ;
			}
		}

}









__global__ void
kernel_push2_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, 
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1)
{
	int x1 = threadIdx.x ;
	int y1 = threadIdx.y ;
	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;

	__shared__ int height_fn[356];


	int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

	height_fn[temp_mult] = g_graph_height[thid] ;

	(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
	(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
	(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
	(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

	__syncthreads();

	int flow_push = 0, min_flow_pushed = 0 ;
	flow_push = g_push_reser[thid] ;

	
		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}else atomicSub(&g_pull_left[thid-1], 1) ;

			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}else atomicSub( &g_pull_down[thid+width1], 1) ;
		}
	
		__syncthreads() ; 
		min_flow_pushed = g_left_weight[thid] ;
		flow_push = g_push_reser[thid] ;

		if(flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
			g_relabel_mask[thid] = 2 ;
		else
		{
			( flow_push > 0 && ( ( (height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && g_left_weight[thid] > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;
		}




		__syncthreads() ;


		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}else atomicSub(&g_pull_left[thid-1], 1) ;

			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}else atomicSub( &g_pull_down[thid+width1], 1) ;
		}
}

__global__ void
kernel_End( int *g_stochastic, int *g_count_blocks, int *g_counter)
{
	int thid = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( thid < ( *g_counter ) )
	{
		if( g_stochastic[thid] == 1 )
			atomicAdd(g_count_blocks,1);
			//(*g_count_blocks) = (*g_count_blocks) + 1 ; 
	}
}


/*__global__ void
kernel_End( int *g_stochastic, int *g_count_blocks, int *g_counter)
{
	int thid = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( thid < ( *g_counter ) )
	{
		if( g_stochastic[thid] == 1 )
			atomicAdd(g_count_blocks,1);
			//(*g_count_blocks) = (*g_count_blocks) + 1 ; 
	}
}
*/



__global__ void
kernel_push1_start_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, 
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish)
{
		int x1 = threadIdx.x ;
		int y1 = threadIdx.y ;
		int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		int thid = __umul24( y , width1 ) + x ;

		__shared__ int height_fn[356];


		int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

		height_fn[temp_mult] = g_graph_height[thid] ;

		(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
		(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
		(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
		(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

		__syncthreads();

		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[thid] ;


		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);

				flow_push = flow_push - min_flow_pushed ;
			}



		}


		__syncthreads() ;
		min_flow_pushed = g_left_weight[thid] ;

			( flow_push > 0 && ( ((height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && min_flow_pushed > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;



		if(thid < graph_size1 && x < width - 1  && x > 0 && y < rows - 1  && y > 0  )
		{
			if(g_sink_weight[thid] > 0)
			{
				g_height_write[thid] = 1 ;
			}
			else
			{
				int min_height = graph_size ;
				(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1] ) ? min_height = height_fn[temp_mult - 1] : 0 ;
				(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0 ;
				(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34] ) ? min_height = height_fn[temp_mult + 34] : 0 ;
				(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34] ) ? min_height = height_fn[temp_mult - 34] : 0 ;
				g_height_write[thid] = min_height + 1 ;
			}
		}

}



__global__ void
kernel_push1_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, /*int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,*/
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_stochastic,int *g_block_num )
{
	if(d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] == 1 )
	{
		int x1 = threadIdx.x ;
		int y1 = threadIdx.y ;
		int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		int thid = __umul24( y , width1 ) + x ;

		__shared__ int height_fn[356];
		
		int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

		height_fn[temp_mult] = g_graph_height[thid] ;

		(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
		(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
		(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
		(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

		__syncthreads();

		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[thid] ;


		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}//else atomicSub(&g_pull_left[thid-1], 1) ;



			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} //else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}//else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}//else atomicSub( &g_pull_down[thid+width1], 1) ;
		}
	}

}



__global__ void
kernel_push2_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish, int *g_block_num)
{
	if(d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] == 1 )
	{
		int x1 = threadIdx.x ;
		int y1 = threadIdx.y ;
		int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		int thid = __umul24( y , width1 ) + x ;

		__shared__ int height_fn[356];


		int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

		height_fn[temp_mult] = g_graph_height[thid] ;

		(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
		(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
		(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
		(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

		__syncthreads();

		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[thid] ;


		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}else atomicSub(&g_pull_left[thid-1], 1) ;



			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}else atomicSub( &g_pull_down[thid+width1], 1) ;
		}
			
			
		__syncthreads() ; 
		min_flow_pushed = g_left_weight[thid] ;
		flow_push = g_push_reser[thid] ;

		if(flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
			g_relabel_mask[thid] = 2 ;
		else
		{
			( flow_push > 0 && ( ( (height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && g_left_weight[thid] > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;
		}




		__syncthreads() ;




		if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
		{
			int temp_weight = 0;


			temp_weight = g_sink_weight[thid] ;
			min_flow_pushed = flow_push ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[thid] = temp_weight ;
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
			}


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;

			temp_weight = g_left_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[thid] , min_flow_pushed);
				atomicAdd(&g_right_weight[thid-1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-1], min_flow_pushed);

			}else atomicSub(&g_pull_left[thid-1], 1) ;



			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[thid] , min_flow_pushed);
				atomicAdd(&g_down_weight[thid-width1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);

			} else atomicSub(&g_pull_up[thid - width1] , 1) ;


			flow_push = g_push_reser[thid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[thid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 )
			{
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[thid] , min_flow_pushed);
				atomicAdd(&g_left_weight[thid+1],min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
			}else atomicSub( &g_pull_right[thid + 1], 1) ;


			flow_push = g_push_reser[thid] ;

			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[thid] ;


			if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 )
			{
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[thid] , min_flow_pushed);
				atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
				atomicSub(&g_push_reser[thid] , min_flow_pushed);
				atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
			}else atomicSub( &g_pull_down[thid+width1], 1) ;
		}

		

	}
}






__global__ void
kernel_bfs_t(int *g_push_reser, int  *g_sink_weight, int *g_graph_height, bool *g_pixel_mask,
		int vertex_num, int width, int height, int vertex_num1, int width1, int height1)
{

	int thid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x ;

	if(thid < vertex_num && g_pixel_mask[thid] == true )
	{
		int col = thid % width1 , row = thid / width1 ;

		if(col > 0 && row > 0 && col < width - 1 && row < height - 1 && g_push_reser[thid] > 0 )
		{
			g_graph_height[thid] = 1 ;
			g_pixel_mask[thid] = false ;
		}
		else
			if(g_sink_weight[thid] > 0)
			{
				g_graph_height[thid] = -1 ;
				g_pixel_mask[thid] = false ;
			}
	}
}

		

__global__ void
kernel_push_stochastic1( int *g_push_reser, int *s_push_reser, int *g_count_blocks, bool *g_finish, int *g_block_num, int width1)
{
	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;
	
	s_push_reser[thid] = g_push_reser[thid] ;
	
	if( thid == 0 )
	{
		if((*g_count_blocks) == 0 )
			(*g_finish) = false ; 
	}
}


__global__ void
kernel_push_atomic2( int *g_terminate, int *g_push_reser, int *s_push_reser, int *g_block_num, int width1)
{

	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;
	
	if( s_push_reser[thid] - g_push_reser[thid] != 0)
	{
		g_terminate[blockIdx.y * (*g_block_num) + blockIdx.x] = 1 ;
	}

}



__global__ void
kernel_push_stochastic2( int *g_terminate, int *g_relabel_mask, int *g_push_reser, int *s_push_reser, int *d_stochastic, int *g_block_num, int width1)
{

	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;
	
	if( s_push_reser[thid] - g_push_reser[thid] != 0)
	{
		g_terminate[blockIdx.y * (*g_block_num) + blockIdx.x] = 1 ;
	}

	if ( g_relabel_mask[thid] == 1 )
	{
		d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] = 1 ;
	} 

}



__global__ void
kernel_push1_start_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser,
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish)
{
	int x1 = threadIdx.x ;
	int y1 = threadIdx.y ;
	int x  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int y  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int thid = __umul24( y , width1 ) + x ;

	__shared__ int height_fn[356];


	int temp_mult = __umul24(y1+1 , 34 ) + x1 + 1, temp_mult1 = __umul24(y1,32) + x1 ;

	height_fn[temp_mult] = g_graph_height[thid] ;

	(threadIdx.x == 31 && x < width1 - 1 ) ? height_fn[temp_mult + 1] =  (g_graph_height[thid + 1]) : 0;
	(threadIdx.x == 0 && x > 0 ) ? height_fn[temp_mult - 1] = (g_graph_height[thid - 1]) : 0;
	(threadIdx.y == 7 && y < rows1 - 1 ) ? height_fn[temp_mult + 34] = (g_graph_height[thid + width1]) : 0;
	(threadIdx.y == 0 && y > 0 ) ? height_fn[temp_mult - 34] = (g_graph_height[thid - width1]) : 0;

	__syncthreads();

	int flow_push = 0, min_flow_pushed = 0 ;
	flow_push = g_push_reser[thid] ;


	if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
	{
		int temp_weight = 0;


		temp_weight = g_sink_weight[thid] ;
		min_flow_pushed = flow_push ;

		if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1 )
		{
			(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
			temp_weight = temp_weight - min_flow_pushed ;
			g_sink_weight[thid] = temp_weight ;
			atomicSub(&g_push_reser[thid] , min_flow_pushed);

			flow_push = flow_push - min_flow_pushed ;
		}



	}


	__syncthreads() ;
	min_flow_pushed = g_left_weight[thid] ;

	( flow_push > 0 && ( ((height_fn[temp_mult] == height_fn[temp_mult-1] + 1 ) && min_flow_pushed > 0  ) ||( (height_fn[temp_mult] == height_fn[temp_mult+1]+1 ) && g_right_weight[thid] > 0) || ( ( height_fn[temp_mult] == height_fn[temp_mult+34]+1 ) && g_down_weight[thid] > 0) || ( (height_fn[temp_mult] == height_fn[temp_mult-34]+1 ) && g_up_weight[thid] > 0 ) || ( height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0 )  ) ) ? g_relabel_mask[thid] = 1 : g_relabel_mask[thid] = 0 ;



	if(thid < graph_size1 && x < width - 1  && x > 0 && y < rows - 1  && y > 0  )
	{
		if(g_sink_weight[thid] > 0)
		{
			g_height_write[thid] = 1 ;
		}
		else
		{
			int min_height = graph_size ;
			(min_flow_pushed > 0 && min_height > height_fn[temp_mult - 1] ) ? min_height = height_fn[temp_mult - 1] : 0 ;
			(g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1]) ? min_height = height_fn[temp_mult + 1] : 0 ;
			(g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34] ) ? min_height = height_fn[temp_mult + 34] : 0 ;
			(g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34] ) ? min_height = height_fn[temp_mult - 34] : 0 ;
			g_height_write[thid] = min_height + 1 ;
		}
	}

}



__global__ void
kernel_bfs(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_graph_height, bool *g_pixel_mask, int vertex_num,int width,int height,
		int vertex_num1, int width1, int height1, bool *g_over, int *g_counter)
{
	/*******************************
	 *threadId is calculated ******
	 *****************************/

	int thid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x ;

	if(thid < vertex_num && g_pixel_mask[thid] == true)
	{
		int col = thid % width1 , row = thid / width1 ;

		if(col < width - 1 && col > 0 && row < height - 1 && row > 0 )
		{
			int height_l = 0, height_d = 0, height_u = 0 , height_r = 0 ;
			height_r = g_graph_height[thid+1] ;
			height_l = g_graph_height[thid-1] ;
			height_d = g_graph_height[thid+width1] ;
			height_u = g_graph_height[thid-width1] ;

			if(((height_l == (*g_counter) && g_right_weight[thid-1] > 0)) ||((height_d == (*g_counter) && g_up_weight[thid+width1] > 0) || ( height_r == (*g_counter) && g_left_weight[thid+1] > 0 ) || ( height_u == (*g_counter) && g_down_weight[thid-width1] > 0 ) ))
			{
				g_graph_height[thid] = (*g_counter) + 1 ;
				g_pixel_mask[thid] = false ;
				*g_over = true ;
			}
		}
	}
}



/************************************************************
 * functions to construct the graph on the device          **
 * *********************************************************/

__device__
void add_edge(int from, int to, int cap, int rev_cap, int type, int *d_left_weight,
		int *d_right_weight, int *d_down_weight, int *d_up_weight)
{
	if(type==1)
	{
		d_left_weight[from] = d_left_weight[from]+cap;
		d_right_weight[to] = d_right_weight[to]+rev_cap;
	}
	if(type==2)
	{
		d_right_weight[from] = d_right_weight[from]+cap;
		d_left_weight[to] = d_left_weight[to]+rev_cap;
	}
	if(type==3)
	{
		d_down_weight[from] = d_down_weight[from]+cap;
		d_up_weight[to] = d_up_weight[to]+rev_cap;
	}
	if(type==4)
	{
		d_up_weight[from] = d_up_weight[from]+cap;
		d_down_weight[to] = d_down_weight[to]+cap;
	}
}

__device__
void add_tweights(int i, int cap_source, int  cap_sink, int *d_push_reser, int *d_sink_weight)
{
	int diff = cap_source - cap_sink ;

	if(diff>0)
	{
		d_push_reser[i] = d_push_reser[i] + diff ;
	}
	else
	{
		d_sink_weight[i] = d_sink_weight[i] - diff ;
	}
}

__device__
void add_term1(int i, int A, int B, int *d_push_reser, int *d_sink_weight)
{
	add_tweights(i,B,A, d_push_reser, d_sink_weight);
}

__device__
void add_t_links_Cue(int alpha_label, int thid, int *d_left_weight, int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight,
		int *dPixelLabel, int *dDataTerm, int width , int height, int num_labels)
{
	{
		if(dPixelLabel[thid]!=alpha_label) {
			add_term1(thid , dDataTerm[thid*num_labels+alpha_label] , dDataTerm[thid * num_labels + dPixelLabel[thid]], d_push_reser, d_sink_weight  );
		}
	}
}


__device__
void add_t_links(int alpha_label, int thid, int *d_left_weight, int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight,
		int *dPixelLabel, int *dDataTerm, int width , int height, int num_labels)
{
	{
		if(dPixelLabel[thid]!=alpha_label) {
			add_term1(thid , dDataTerm[thid*num_labels+alpha_label] , dDataTerm[thid * num_labels + dPixelLabel[thid]], d_push_reser, d_sink_weight  );
		}
	}
}



__device__
void add_term2(int x, int y, int A, int B, int C, int D, int type, int *d_left_weight,
		int *d_right_weight, int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight  )
{
	if ( A+D > C+B) {
		int delta = A+D-C-B;
		int subtrA = delta/3;

		A = A-subtrA;
		C = C+subtrA;
		B = B+(delta-subtrA*2);
#ifdef COUNT_TRUNCATIONS
		truncCnt++;
#endif
	}
#ifdef COUNT_TRUNCATIONS
	totalCnt++;
#endif

	add_tweights(x, D, A, d_push_reser, d_sink_weight);

	B -= A; C -= D;

	if (B < 0)
	{
		add_tweights(x, 0, B, d_push_reser, d_sink_weight);
		add_tweights(y, 0, -B, d_push_reser, d_sink_weight ) ;
		add_edge(x, y, 0, B+C,type , d_left_weight, d_right_weight, d_down_weight, d_up_weight );
	}
	else if (C < 0)
	{
		add_tweights(x, 0, -C, d_push_reser, d_sink_weight);
		add_tweights(y, 0, C , d_push_reser, d_sink_weight);
		add_edge(x, y, B+C, 0,type , d_left_weight, d_right_weight, d_down_weight, d_up_weight);
	}
	else
	{
		add_edge(x, y, B, C,type, d_left_weight, d_right_weight , d_down_weight, d_up_weight);
	}
}


__device__
void set_up_expansion_energy_G_ARRAY(int alpha_label,int thid, int *d_left_weight,int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser,
		int *d_sink_weight, int *dPixelLabel, int *dDataTerm, int *dSmoothTerm,
		int width , int height, int num_labels )
{
	int x,y,nPix;

	int weight;

	int i = thid ;
	{
		if(dPixelLabel[i]!=alpha_label)
		{
			y = i/width;
			x = i - y*width;

			if ( x < width - 1 )
			{
				nPix = i + 1;
				weight = 1 ;
				if ( dPixelLabel[nPix] != alpha_label )
				{
					add_term2(i,nPix,
							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,

							2, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight); // 1-left, 2-right, 3-down, 4-up
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}

			if ( y < height - 1 )
			{
				nPix = i + width;
				weight = 1 ;
				if ( dPixelLabel[nPix] != alpha_label )
				{
					add_term2(i,nPix,
							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,



							3, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight );
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}
			if ( x > 0 )
			{
				nPix = i - 1;
				weight = 1 ;
				if ( dPixelLabel[nPix] == alpha_label )
					add_term1(i,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
							( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
							d_push_reser, d_sink_weight );
			}

			if ( y > 0 )
			{
				nPix = i - width;
				weight = 1 ;
				if ( dPixelLabel[nPix] == alpha_label )
				{
					add_term1(i,

							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
							d_push_reser, d_sink_weight);
				}
			}
		}
	}
}



__device__
void set_up_expansion_energy_G_ARRAY_Cue(int alpha_label,int thid, int *d_left_weight,int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser,
		int *d_sink_weight, int *dPixelLabel, int *dDataTerm, int *dSmoothTerm,
		int *dHcue, int *dVcue, int width , int height, int num_labels )
{
	int x,y,nPix;

	int weight;

	int i = thid ;
	{
		if(dPixelLabel[i]!=alpha_label)
		{
			y = i/width;
			x = i - y*width;

			if ( x < width - 1 )
			{
				nPix = i + 1;
				weight=dHcue[i];
				if ( dPixelLabel[nPix] != alpha_label )
				{
					add_term2(i,nPix,
							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,

							2, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight); // 1-left, 2-right, 3-down, 4-up
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}


			if ( y < height - 1 )
			{
				nPix = i + width;
				weight=dVcue[i];
				if ( dPixelLabel[nPix] != alpha_label )
				{
					add_term2(i,nPix,
							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
							( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,



							3, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight );
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}
			if ( x > 0 )
			{
				nPix = i - 1;
				weight=dHcue[nPix];
				if ( dPixelLabel[nPix] == alpha_label )
					add_term1(i,
							( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
							( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
							d_push_reser, d_sink_weight );
			}

			if ( y > 0 )
			{
				nPix = i - width;
				weight = dVcue[nPix] ;
				if ( dPixelLabel[nPix] == alpha_label )
				{
					add_term1(i,

							( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
							( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
							d_push_reser, d_sink_weight);
				}
			}
		}
	}
}



__global__
void CudaWeightCue(int alpha_label, int *d_left_weight, int *d_right_weight, int *d_down_weight,
		int *d_up_weight, int *d_push_reser, int *d_sink_weight, int *dPixelLabel,
		int *dDataTerm, int *dSmoothTerm, int *dHcue, int *dVcue, int width, int height, int num_labels )
{
	int thid = blockIdx.x * 256 + threadIdx.x ;

	add_t_links_Cue(alpha_label, thid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, width, height, num_labels);

	set_up_expansion_energy_G_ARRAY_Cue(alpha_label, thid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, dSmoothTerm, dHcue, dVcue, width, height, num_labels);
}


__global__
void CudaWeight(int alpha_label, int *d_left_weight, int *d_right_weight, int *d_down_weight,
		int *d_up_weight, int *d_push_reser, int *d_sink_weight, int *dPixelLabel,
		int *dDataTerm, int *dSmoothTerm, int width, int height, int num_labels)
{
	int thid = blockIdx.x * 256 + threadIdx.x ;

	add_t_links(alpha_label, thid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, width, height, num_labels);

	set_up_expansion_energy_G_ARRAY(alpha_label, thid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, dSmoothTerm, width, height, num_labels);
}



/*********************************************************
 * function which adjusts the array size for efficiency **
 * consideration                                        **
 * ******************************************************/

__global__
void adjustedgeweight(int *d_left_weight, int *d_right_weight, int *d_down_weight, int *d_up_weight,
		int *d_push_reser, int *d_sink_weight, int *temp_left_weight, int *temp_right_weight,
		int *temp_down_weight, int *temp_up_weight, int *temp_push_reser, int *temp_sink_weight,
		int width, int height, int graph_size, int width1, int height1, int graph_size1)
{
	int thid = blockIdx.x * 256 + threadIdx.x ;

	if( thid < graph_size1 )
	{
		int row = thid / width1 , col = thid % width1 ;

		if(row < height && col < width)
		{
			temp_left_weight[row* width1 + col] = d_left_weight[row * width + col] ;
			temp_right_weight[row * width1 + col] = d_right_weight[row * width + col] ;
			temp_down_weight[row * width1 + col] = d_down_weight[row * width + col] ;
			temp_up_weight[row * width1 + col] = d_up_weight[row * width + col] ;
			temp_push_reser[row * width1 + col] = d_push_reser[row * width + col] ;
			temp_sink_weight[row * width1 + col] = d_sink_weight[row * width + col] ;

		}
		else
		{
			temp_left_weight[row * width1 + col] = 0 ;
			temp_right_weight[row * width1 + col] = 0 ;
			temp_down_weight[row * width1 + col] = 0 ;
			temp_up_weight[row * width1 + col] = 0 ;
			temp_push_reser[row * width1 + col] = 0 ;
			temp_sink_weight[row * width1 + col] = 0 ;
		}
	}
}



/************************************************************
 * Intializes memory on the gpu                            **
 * ********************************************************/

__global__
void copyedgeweight( int *d_left_weight, int *d_right_weight, int *d_down_weight, int *d_up_weight,
		int *d_push_reser, int *d_sink_weight, int *temp_left_weight, int *temp_right_weight,
		int *temp_down_weight, int *temp_up_weight, int *temp_push_reser, int *temp_sink_weight,
		int *d_pull_left, int *d_pull_right, int *d_pull_down, int *d_pull_up, int *d_relabel_mask,
		int *d_graph_heightr, int *d_graph_heightw, int width, int height, int graph_size, int width1, int height1, int graph_size1)
{
	int thid = blockIdx.x * 256 + threadIdx.x ;

	if( thid < graph_size1 )
	{
		d_left_weight[thid] = temp_left_weight[thid] ;
		d_right_weight[thid] = temp_right_weight[thid] ;
		d_down_weight[thid] = temp_down_weight[thid] ;
		d_up_weight[thid] = temp_up_weight[thid] ;
		d_push_reser[thid] = temp_push_reser[thid] ;
		d_sink_weight[thid] = temp_sink_weight[thid] ;

		d_pull_left[thid] = 0 ;
		d_pull_right[thid] = 0 ;
		d_pull_down[thid] = 0 ;
		d_pull_up[thid] = 0 ;
		d_relabel_mask[thid] = 0 ;
		d_graph_heightr[thid] = 1 ;
		d_graph_heightw[thid] = 1 ;

	}
}




#endif
