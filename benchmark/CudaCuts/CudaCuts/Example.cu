#include "hip/hip_runtime.h"

/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       ** 
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **  
*                                                                                          **   
* Copyright (c) 2008 International Institute of Information Technology.                    **  
* All rights reserved.                                                                     **  
*                                                                                          ** 
* Permission to use, copy, modify and distribute this software and its documentation for   ** 
* educational purpose is hereby granted without fee, provided that the above copyright     ** 
* notice and this permission notice appear in all copies of this software and that you do  **
* not sell the software.                                                                   **  
*                                                                                          **
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
* OTHERWISE.                                                                               **  
*                                                                                          **
* Created By Vibhav Vineet.                                                                ** 

********************************************************************************************/

#define CUT_EXIT(argc, argv) \
    if (!checkCmdLineFlag(argc, (const char**)argv, "noprompt")) { \
        printf("\nPress ENTER to exit...\n"); \
        fflush( stdout); \
        fflush( stderr); \
        getchar(); \
    } \
    exit(EXIT_SUCCESS);

#include "CudaCuts.cu"
#include "Example.h"

#include <string.h>

#include "cl/cl.h"

#include "QIO.h"
#include "QUtility.h"

using namespace std; 

int main(int argc,char** argv)
{
	load_files(argv[1]) ;

	int initCheck = cudaCutsInit(width, height ,num_Labels) ;
	
	printf("Compute Capability %d\n",initCheck);

	if( initCheck > 0 )
	{
		printf("The grid is initialized successfully\n");
	}
	else 
		if( initCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int dataCheck   =  cudaCutsSetupDataTerm( dataTerm );

	if( dataCheck == 0 )
	{
		printf("The dataterm is set properly\n");
		
	}
	else 
		if( dataCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int smoothCheck =  cudaCutsSetupSmoothTerm( smoothTerm );

	if( smoothCheck == 0 )
	{
		printf("The smoothnessterm is set properly\n");
	}
	else
		if( smoothCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int hcueCheck   =  cudaCutsSetupHCue( hCue );

	if( hcueCheck == 0 )
	{
		printf("The HCue is set properly\n");
	}
	else
		if( hcueCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int vcueCheck   =  cudaCutsSetupVCue( vCue );

	if( vcueCheck == 0 )
	{
		printf("The VCue is set properly\n");
	}
	else 
		if( vcueCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int graphCheck = cudaCutsSetupGraph();

	if( graphCheck == 0 )
	{
		printf("The graph is constructed successfully\n");
	}
	else 
		if( graphCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int optimizeCheck = -1; 
	if( initCheck == 1 )
	{
		//CudaCuts involving atomic operations are called
		//optimizeCheck = cudaCutsAtomicOptimize();
		//CudaCuts involving stochastic operations are called
		optimizeCheck = cudaCutsStochasticOptimize();
	}

	if( optimizeCheck == 0 )
	{
		printf("The algorithm successfully converged\n");
	}
	else 
		if( optimizeCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int resultCheck = cudaCutsGetResult( );

	if( resultCheck == 0 )
	{
		printf("The pixel labels are successfully stored\n");
	}
	else 
		if( resultCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}
		
	int energy = cudaCutsGetEnergy(); 
    printf("TOTAL ENERGY: %d\n",energy);

	initFinalImage();
	
	cudaCutsFreeMem();
	
	exit(1);
	CUT_EXIT(argc,argv);
}

bool parseDataFile(const std::string& name, std::string& objectFileName, cl_uint4& volumeSize, cl_ulong& maxFLow)
{
    std::string dataFileContent, line;
    if (!QIO::getFileContent(name, dataFileContent)) return false;

    std::stringstream data(dataFileContent, std::stringstream::in);
    bool error = false;
    ::size_t position = std::string::npos;
    while (!data.eof())
    {
        getline(data, line);
        std::stringstream buffer(std::stringstream::in | std::stringstream::out);
        if ((position = line.find("ObjectFileName")) != std::string::npos)
        {
            if ((position = line.find(':')) == std::string::npos)
            {
                error = true;
                break;
            }
            objectFileName = line.substr(position + 1);
            QUtility::trim(objectFileName);
        }
        else if ((position = line.find("Resolution")) != std::string::npos)
        {
            if ((position = line.find(':')) == std::string::npos)
            {
                error = true;
                break;
            }
            buffer << line.substr(position + 1);
            unsigned int x = 0, y = 0, z = 0;
            buffer >> x >> y >> z;
            if (x <= 0 || y <= 0 || z <= 0)
            {
                error = true;
                break;
            }
            volumeSize.s[0] = x;
            volumeSize.s[1] = y;
            volumeSize.s[2] = z;
        }
        else if ((position = line.find("MaxFlow")) != std::string::npos)
        {
            if ((position = line.find(':')) == std::string::npos)
            {
                error = true;
                break;
            }
            buffer << line.substr(position + 1);
            int flow = 0;
            buffer >> flow;
            if (flow < 0)
            {
                error = true;
                break;
            }

            maxFLow = flow;
        }
        else
        {
            std::cerr << " > WARNING: skipping line \"" << line << "\"." << std::endl;
        }
    }

    if (error)
    {
        std::cerr << " > ERROR: parsing \"" << line << "\"." << std::endl;
        return false;
    }

    return true;
}

void load_files(char *filename)
{
    /**
	LoadDataFile(filename, width, height, num_Labels, dataTerm, smoothTerm, hCue, vCue);
    /*/
    LoadDataFileBenchmark(filename, width, height, num_Labels, dataTerm, smoothTerm, hCue, vCue);
    //*/
}

void initFinalImage()
{
	out_pixel_values=(int**)malloc(sizeof(int*)*height);

	for(int i = 0 ; i < height ; i++ )
	{
		out_pixel_values[i] = (int*)malloc(sizeof(int) * width ) ;
		for(int j = 0 ; j < width ; j++ ) {
			out_pixel_values[i][j]=0;
		}
	}

	writeImage() ;
}

void writeImage()
{
	for(int i = 0 ; i <  graph_size1 ; i++)
	{

		int row = i / width1, col = i % width1 ;

		if(row >= 0 && col >= 0 && row <= height -1 && col <= width - 1 )
			out_pixel_values[row][col]=pixelLabel[i]*255;
	}

	write_image();
}

void write_image()
{

	FILE* fp=fopen("result_sponge/flower_cuda_test.pgm","w");

	fprintf(fp,"%c",'P');
	fprintf(fp,"%c",'2');
	fprintf(fp,"%c",'\n');
	fprintf(fp,"%d %c %d %c ",width,' ',height,'\n');
	fprintf(fp,"%d %c",255,'\n');

	for(int i=0;i<height;i++)
	{
		for(int j=0;j<width;j++)
		{
			fprintf(fp,"%d\n",out_pixel_values[i][j]);
		}
	}
	fclose(fp);
}

void LoadDataFile(char *filename, int &width, int &height, int &nLabels,
		int *&dataCostArray,
		int *&smoothCostArray,
		int *&hCue,
		int *&vCue)
{
	printf("enterd\n");
	
	FILE *fp = fopen(filename,"r");
	
	fscanf(fp,"%d %d %d",&width,&height,&nLabels);
    
	int i, n, x, y;
	int gt;
	for(i = 0; i < width * height; i++)
		fscanf(fp,"%d",&gt);
    
	dataCostArray = new int[width * height * nLabels];
	for(int c=0; c < nLabels; c++) {
		n = c;
		for(i = 0; i < width * height; i++) {
			fscanf(fp,"%d",&dataCostArray[n]);
			n += nLabels;
		}
	}
    
	hCue = new int[width * height];
	vCue = new int[width * height];

	n = 0;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width-1; x++) {
			fscanf(fp,"%d",&hCue[n++]);
		}
		hCue[n++] = 0;
	}

	n = 0;
	for(y = 0; y < height-1; y++) {
		for(x = 0; x < width; x++) {
			fscanf(fp,"%d",&vCue[n++]);
		}
	}
	for(x = 0; x < width; x++) {
		vCue[n++] = 0;
	}
    
	fclose(fp);
	smoothCostArray = new int[nLabels * nLabels];

	smoothCostArray[0] = 0 ;
	smoothCostArray[1] = 1 ;
	smoothCostArray[2] = 1 ;
	smoothCostArray[3] = 0 ;
}

void LoadDataFileBenchmark(char *filename, int &width, int &height, int &nLabels,
		int *&dataCostArray,
		int *&smoothCostArray,
		int *&hCue,
		int *&vCue)
{
    std::string original = "original/";
    std::string scaled = "scaled/";
    std::string scaled2 = "scaled-2/";
    std::string flower = "flower/normalized_flower.raw.dat";
    std::string person = "person/normalized_person.raw.dat";
    std::string sponge = "sponge/normalized_sponge.raw.dat";

    std::string path = "F:/88Datasets/Image Data/";
    std::string type = scaled2;
    std::string instance = flower;
    std::string dataFileName(path + type + instance);
    int position = dataFileName.find_last_of("\\");
    if (position == std::string::npos) position = dataFileName.find_last_of("/");
    if (position == std::string::npos) position = dataFileName.size() - 1;
    std::string dataFilePath = dataFileName.substr(0, position + 1);

    std::string objectFileName;
    cl_uint4 volumeSize;
    cl_ulong maxFlow;
    parseDataFile(dataFileName, objectFileName, volumeSize, maxFlow);

    std::string nodeFileExtension = ".node";
    cl_uint4 clTileSize2D = { 16, 16,  1,  1 };
    cl_uint4 nodeSize =
    {
        QUtility::ceil(clTileSize2D.s[0], volumeSize.s[0]),
        QUtility::ceil(clTileSize2D.s[1], volumeSize.s[1]),
        1,
        1
    };

    ::size_t size = nodeSize.s[0] * nodeSize.s[1];
    std::vector<cl_node_2d> clNode(size);
    if (!QIO::getFileData(dataFilePath + objectFileName + nodeFileExtension, clNode.data(), clNode.size() * sizeof(cl_node_2d))) return;
    
    printf("enterd\n");
	
    width = volumeSize.s[0];
    height = volumeSize.s[1];
    nLabels = 2;
	
	dataCostArray = new int[width * height * nLabels];
	hCue = new int[width * height];
	vCue = new int[width * height];
    for (int y = 0; y < height; y++)
        for (int x = 0; x < width; x++)
        {
            int index = x + width * y;
            cl_node_2d* node = clNode.data() + x + nodeSize.s[0] * y;
            if (node->excessFlow > 0)
            {
                dataCostArray[index * nLabels + 0] = node->excessFlow;
                dataCostArray[index * nLabels + 1] = 0;
            }
            else
            {
                dataCostArray[index * nLabels + 0] = 0;
                dataCostArray[index * nLabels + 1] = -node->excessFlow;
            }

            hCue[index] = node->capacity[0];
            vCue[index] = node->capacity[1];
        };

	smoothCostArray = new int[nLabels * nLabels];

	smoothCostArray[0] = 0 ;
	smoothCostArray[1] = 1 ;
	smoothCostArray[2] = 1 ;
	smoothCostArray[3] = 0 ;
}