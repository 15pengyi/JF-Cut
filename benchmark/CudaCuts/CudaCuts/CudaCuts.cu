#include "hip/hip_runtime.h"



/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       ** 
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **  
*                                                                                          **   
* Copyright (c) 2008 International Institute of Information Technology.                    **  
* All rights reserved.                                                                     **  
*                                                                                          ** 
* Permission to use, copy, modify and distribute this software and its documentation for   ** 
* educational purpose is hereby granted without fee, provided that the above copyright     ** 
* notice and this permission notice appear in all copies of this software and that you do  **
* not sell the software.                                                                   **  
*                                                                                          **
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
* OTHERWISE.                                                                               **  
*                                                                                          **
* Created By Vibhav Vineet.                                                                ** 
********************************************************************************************/


#ifndef _CUDACUTS_CU_
#define _CUDACUTS_CU_

#include "CudaCuts.h"

/********************************************************************
 * cudaCutsInit(width, height, numOfLabels) function sets the      **
 * width, height and numOfLabels of grid. It also initializes the  **
 * block size  on the device and finds the total number of blocks  **
 * running in parallel on the device. It calls checkDevice         **
 * function which checks whether CUDA compatible device is present **
 * on the system or not. It allocates the memory on the host and   **
 * the device for the arrays which are required through the        **
 * function call h_mem_init and segment_init respectively. This    **
 * function returns 0 on success or -1 on failure if there is no   **
 * * * CUDA compatible device is present on the system             **
 * *****************************************************************/

int cudaCutsInit(int widthGrid, int heightGrid, int labels)
{
	deviceCount = checkDevice() ; 	
	
	printf("No. of devices %d\n",deviceCount);
	if( deviceCount < 1 )
		return -1 ; 
	
	int cuda_device = 0;
	
	hipSetDevice( cuda_device );

	hipDeviceProp_t device_properties;

	checkCudaErrors( hipGetDeviceProperties(&device_properties, cuda_device) );
	
    if(1 < device_properties.major)
        deviceCheck = 1 ; 
	else if( (1 == device_properties.major) && (device_properties.minor < 1))
		deviceCheck = 2 ; 
	else if( (1 == device_properties.major) && (device_properties.minor >= 1))
		deviceCheck = 1 ; 
	else 
		deviceCheck = 0 ; 

	width =  widthGrid ; 
	height = heightGrid ; 
	num_Labels = labels ; 

	blocks_x = 1 ;
	blocks_y = 1 ;
	num_of_blocks = 1 ; 

	num_of_threads_per_block = 256 ; 
	threads_x = 32 ; 
	threads_y = 8 ;  

	width1 = threads_x * ((int)ceil((float)width/ (float)threads_x))  ;  
	height1 = threads_y * ( (int)ceil((float) height / (float) threads_y ))   ; 

	graph_size = width * height ; 
	graph_size1 = width1 * height1 ; 
	size_int = sizeof(int) * graph_size1 ; 

	blocks_x = (int)((ceil)((float)width1/(float)threads_x));
	blocks_y = (int)((ceil)((float)height1/(float)threads_y));

	num_of_blocks = (int)((ceil)((float)graph_size1/(float)num_of_threads_per_block));

	h_mem_init(); 
	d_mem_init(); 
	cueValues = 0 ;

	return deviceCheck ; 
}

int checkDevice()
{
	int deviceCount ;

	hipGetDeviceCount( &deviceCount ) ;

	if( deviceCount == 0 )
	{
		return -1;
	}

	return deviceCount ;
}

void h_mem_init()
{
	h_reset_mem  =  ( int* )malloc(sizeof(int) * graph_size1);
	h_graph_height  =  (int* )malloc(size_int);
	pixelLabel  =  (int*)malloc( size_int);
	h_pixel_mask = (bool*)malloc( sizeof(bool) * graph_size1  ) ;

	for( int i = 0 ; i < graph_size1 ; i++ )
	{
		pixelLabel[i] = 0 ;
		h_graph_height[i] = 0 ;
	}

	for( int i = 0 ; i < graph_size1 ; i++ )
	{
		h_reset_mem[i] = 0 ;
	}
}

void d_mem_init()
{
	checkCudaErrors( hipMalloc((void**)&d_left_weight, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_right_weight, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_down_weight, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_up_weight, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_push_reser, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );

	//checkCudaErrors( hipMalloc((void**)&s_left_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&s_right_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&s_down_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&s_up_weight, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&s_push_reser, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&s_sink_weight, sizeof(int) * graph_size1 ) );

	checkCudaErrors( hipMalloc((void**)&d_stochastic, sizeof(int) * num_of_blocks ) );
	checkCudaErrors( hipMalloc((void**)&d_stochastic_pixel, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_terminate, sizeof(int) * num_of_blocks ) );

	//checkCudaErrors( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );

	//checkCudaErrors( hipMalloc((void**)&d_pull_left, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_pull_right, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_pull_down, sizeof(int) * graph_size1 ) );
	//checkCudaErrors( hipMalloc((void**)&d_pull_up, sizeof(int) * graph_size1 ) );

	checkCudaErrors( hipMalloc((void**)&d_graph_heightr, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_graph_heightw, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_relabel_mask, sizeof(int) * graph_size1 ) );

	checkCudaErrors( hipMalloc( ( void**)&d_pixel_mask, sizeof(bool)*graph_size1 ) );
	checkCudaErrors( hipMalloc( ( void**)&d_over, sizeof(bool)*1 ) );
	checkCudaErrors(hipMalloc((void**)&d_counter,sizeof(int)));

	checkCudaErrors( hipMalloc( ( void **)&dPixelLabel, sizeof(int) * width1 * height1 ));
	checkCudaErrors( hipMemcpy( d_left_weight, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_right_weight, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_down_weight, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_up_weight, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_push_reser, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_sink_weight, h_reset_mem, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;
	
	checkCudaErrors( hipMalloc((void**)&d_pull_left, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_pull_right, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_pull_down, sizeof(int) * graph_size1 ) );
	checkCudaErrors( hipMalloc((void**)&d_pull_up, sizeof(int) * graph_size1 ) );

	h_relabel_mask = (int*)malloc(sizeof(int)*width1*height1);
	
	     h_stochastic = (int *)malloc(sizeof(int) * num_of_blocks);
	     h_stochastic_pixel = (int *)malloc(sizeof(int) * graph_size1);

	for(int i = 0 ; i < graph_size1 ; i++ )
		h_relabel_mask[i] = 1 ; 
	
	checkCudaErrors( hipMemcpy( d_relabel_mask, h_relabel_mask, sizeof( int ) * graph_size1 , hipMemcpyHostToDevice   ) ) ;

	int *dpixlab = (int*)malloc(sizeof(int)*width1*height1);

	for( int i = 0 ; i < width1 * height1 ; i++ )
	{
		dpixlab[i] = 0 ;
		h_stochastic_pixel[i] = 1 ; 
	}

	for(int i = 0 ; i < num_of_blocks ; i++ )
	{
		h_stochastic[i] = 1 ; 
	}

	checkCudaErrors( hipMemcpy( d_stochastic, h_stochastic, sizeof(int) * num_of_blocks , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_terminate, h_stochastic, sizeof(int) * num_of_blocks , hipMemcpyHostToDevice   ) ) ;
	checkCudaErrors( hipMemcpy( d_stochastic_pixel, h_stochastic_pixel, sizeof(int) * graph_size1 , hipMemcpyHostToDevice   ) ) ;

	checkCudaErrors( hipMemcpy( dPixelLabel, dpixlab, sizeof(int) * width1 * height1 , hipMemcpyHostToDevice   ) ) ;
}

int cudaCutsSetupDataTerm( int *dataTerm )
{
	if( deviceCheck < 1 )
		return -1 ; 

	datacost  =  (int*)malloc(sizeof(int) * width *height * num_Labels );
	
	checkCudaErrors( hipMalloc( ( void **)&dDataTerm, sizeof(int) * width * height * num_Labels ));
	
	checkCudaErrors( hipMemcpy( dDataTerm, dataTerm, sizeof(int) * width * height * num_Labels , hipMemcpyHostToDevice   ) ) ;
	
	for( int i = 0 ; i < width * height * num_Labels ; i++)
	{
		datacost[i] = dataTerm[i] ; 
	}

	return 0 ; 
}


int cudaCutsSetupSmoothTerm( int *smoothTerm )
{
	if( deviceCheck < 1 )
		return -1 ; 

	smoothnesscost  =  (int*)malloc(sizeof(int) * num_Labels * num_Labels );

	checkCudaErrors( hipMalloc( ( void **)&dSmoothTerm, sizeof(int) * num_Labels * num_Labels ));

	checkCudaErrors( hipMemcpy( dSmoothTerm, smoothTerm, sizeof(int) * num_Labels * num_Labels , hipMemcpyHostToDevice   ) ) ;
	
	for( int i = 0 ; i < num_Labels * num_Labels ; i++)
	{
		smoothnesscost[i] = smoothTerm[i] ; 
	}

	return 0 ; 
}

int cudaCutsSetupHCue( int *hCue )
{
	if( deviceCheck < 1 )
		return -1 ; 

	hcue  =  (int*)malloc(sizeof(int) * width * height );

	checkCudaErrors( hipMalloc( ( void **)&dHcue, sizeof(int) * width * height ));

	checkCudaErrors( hipMemcpy( dHcue, hCue, sizeof(int) * width * height , hipMemcpyHostToDevice   ) ) ;

	for( int i = 0 ; i < width * height ; i++)
	{
		hcue[i] = hCue[i] ; 
	}

	cueValues = 1 ; 

	return 0 ; 
}

int cudaCutsSetupVCue( int *vCue )
{
	if( deviceCheck < 1 )
		return -1 ; 

	vcue  =  (int*)malloc(sizeof(int) * width * height );

	checkCudaErrors( hipMalloc( ( void **)&dVcue, sizeof(int) * width * height ));

	checkCudaErrors( hipMemcpy( dVcue, vCue, sizeof(int) * width * height , hipMemcpyHostToDevice   ) ) ;

	for( int i = 0 ; i < width * height ; i++)
	{
		vcue[i] = vCue[i] ; 
	}

	return 0 ; 
}


int cudaCutsSetupGraph( )
{
	if( deviceCheck < 1 )
		return -1 ; 

	int alpha_label = 1 ;

	for( int i = 0 ; i < graph_size1 ; i++ )
	{
		h_reset_mem[i] = 0 ;
		h_graph_height[i] = 0 ;
	}

	int blockEdge = (int)((ceil)((float)( width * height )/ ( float ) 256 ));
	dim3 block_weight(256, 1, 1);
	dim3 grid_weight(blockEdge,1,1);
	
	if( cueValues == 1 )
	{
		CudaWeightCue<<< grid_weight , block_weight >>>( alpha_label, d_left_weight, d_right_weight, d_down_weight, 
								d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, 
								dSmoothTerm, dHcue, dVcue, width, height, 2) ;
	}
    else 
	{
		CudaWeight<<< grid_weight , block_weight >>>( alpha_label, d_left_weight, d_right_weight, d_down_weight, 
								d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, 
								dSmoothTerm, width, height, 2) ;
	}

	int *temp_left_weight, *temp_right_weight, *temp_down_weight, *temp_up_weight, *temp_source_weight, *temp_terminal_weight ;

	checkCudaErrors( hipMalloc( ( void **)&temp_left_weight, sizeof( int ) * graph_size1 ) ) ;
	checkCudaErrors( hipMalloc( ( void **)&temp_right_weight, sizeof( int ) * graph_size1 ) ) ;
	checkCudaErrors( hipMalloc( ( void **)&temp_down_weight, sizeof( int ) * graph_size1 ) ) ;
	checkCudaErrors( hipMalloc( ( void **)&temp_up_weight, sizeof( int ) * graph_size1 ) ) ;
	checkCudaErrors( hipMalloc( ( void **)&temp_source_weight, sizeof( int ) * graph_size1 ) ) ;
	checkCudaErrors( hipMalloc( ( void **)&temp_terminal_weight, sizeof( int ) * graph_size1 ) ) ;

	int blockEdge1 = (int)((ceil)((float)( width1 * height1 )/ ( float ) 256 ));
	dim3 block_weight1(256, 1, 1);
	dim3 grid_weight1(blockEdge1,1,1);

	adjustedgeweight<<<grid_weight1, block_weight1>>>(d_left_weight,d_right_weight,d_down_weight,d_up_weight,d_push_reser,
							d_sink_weight,temp_left_weight,temp_right_weight,temp_down_weight,temp_up_weight,
							temp_source_weight, temp_terminal_weight, width,  height,  graph_size,  width1, 
							height1, graph_size1) ;
	
	copyedgeweight<<<grid_weight1, block_weight1>>>(d_left_weight,d_right_weight,d_down_weight,d_up_weight,d_push_reser,d_sink_weight,
							temp_left_weight,temp_right_weight,temp_down_weight,temp_up_weight,temp_source_weight,
							temp_terminal_weight, d_pull_left, d_pull_right, d_pull_down, d_pull_up, d_relabel_mask, 
							d_graph_heightr, d_graph_heightw, width,  height,  graph_size,  width1, height1, graph_size1) ;

	return 0 ; 
}

int cudaCutsAtomicOptimize(  )
{
	if( deviceCheck < 1 )
	{
		return -1 ; 
	}

	cudaCutsAtomic();

	bfsLabeling( );

	return 0 ; 
}

int cudaCutsStochasticOptimize()
{
	if( deviceCheck < 1 )
	{
		return -1 ; 
	}

	cudaCutsStochastic();

	bfsLabeling( );

	return 0 ; 
}

void cudaCutsAtomic()
{
	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);

	bool finish = true ;
	
	counter = num_of_blocks ;

	int numThreadsEnd = 256, numBlocksEnd = 1 ;
	if( numThreadsEnd > counter)
	{
		numBlocksEnd = 1 ;
		numThreadsEnd = counter ;
	}
	else
	{
		numBlocksEnd = (int)ceil(counter/(double)numThreadsEnd);
	}

	dim3 End_block(numThreadsEnd,1,1);
	dim3 End_grid(numBlocksEnd,1,1);

	int *d_counter ; 
	
	bool *d_finish ; 
	for(int i = 0 ; i < num_of_blocks ; i++ )
	{
		h_stochastic[i] = 0 ; 
	}

	checkCudaErrors( hipMalloc((void**)&d_counter, sizeof(int)));
	checkCudaErrors( hipMalloc((void**)&d_finish, sizeof(bool)));

	checkCudaErrors( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	
	counter = 0 ;
	int *d_relabel ; 

	checkCudaErrors( hipMalloc((void**)&d_relabel,sizeof(int) ));
	
	int h_relabel = 0 ;
	
	int block_num = width1 / 32 ; 

	int *d_block_num; 
	
	checkCudaErrors( hipMalloc((void**)&d_block_num, sizeof(int)));
	checkCudaErrors( hipMemcpy( d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));
	
	int h_count_blocks = num_of_blocks ; 
	int *d_count_blocks; 
	
	checkCudaErrors( hipMalloc((void**)&d_count_blocks, sizeof(int)));
	checkCudaErrors( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
	
	h_count_blocks = 0 ; 

	checkCudaErrors( hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));
	
	counter = 1 ; 
	kernel_push1_start_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
			d_sink_weight, d_push_reser,
			d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
			graph_size1, width1 , height1,d_relabel, d_stochastic, d_counter, d_finish );
	
	int h_terminate_condition = 0 ;
	checkCudaErrors(hipDeviceSynchronize());
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	
	do
	{

		if(counter%10 == 0  )
		{
			finish = true ;
			checkCudaErrors( hipMemcpy( d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1<<<grid_push,block_push>>>(d_push_reser, s_push_reser,  d_count_blocks, d_finish, d_block_num, width1);
			checkCudaErrors( hipMemcpy( &finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
			if ( finish == false ) 
				h_terminate_condition++ ; 
		}
		if(counter%11 == 0 )
		{
			checkCudaErrors(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0 ;
			checkCudaErrors( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_atomic2<<<grid_push,block_push>>>(d_terminate, d_push_reser, s_push_reser, d_block_num, width1);

			kernel_End<<<End_grid, End_block>>>(d_terminate, d_count_blocks, d_counter);

		}
		
		if( counter % 2 == 0 )
		{
				
				kernel_push1_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
						d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
						d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
						graph_size1, width1 , height1);
				
				/*kernel_push2_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
						d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
						d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
						graph_size1, width1 , height1 );
				*/
				kernel_relabel_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
						d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
						d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
						graph_size1, width1 , height1);
		}
		else
		{
				kernel_push1_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
						d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up, 
						d_relabel_mask,d_graph_heightw,d_graph_heightr, graph_size,width,height, 
						graph_size1, width1 , height1);
				
				/*kernel_push2_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
					d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
					d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
					graph_size1, width1 , height1);
				*/	
				kernel_relabel_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
						d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up, 
						d_relabel_mask,d_graph_heightw,d_graph_heightr, graph_size,width,height, 
						graph_size1, width1 , height1);

		}
		counter++ ;
	}
	while( h_terminate_condition != 2);

	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	printf("TT Cuts :: %f\n",sdkGetTimerValue(&timer));
	sdkDeleteTimer(&timer);
}

void cudaCutsStochastic()
{
	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);

	bool finish = true ;

	counter = num_of_blocks ;
	
	int numThreadsEnd = 256, numBlocksEnd = 1 ; 
	if( numThreadsEnd > counter)
	{
		numBlocksEnd = 1 ; 
		numThreadsEnd = counter ; 
	}
	else 
	{
		numBlocksEnd = (int)ceil(counter/(double)numThreadsEnd);
	}
	
	dim3 End_block(numThreadsEnd,1,1);
	dim3 End_grid(numBlocksEnd,1,1);
	
	bool *d_finish ; 
	for(int i = 0 ; i < num_of_blocks ; i++ )
	{
		h_stochastic[i] = 0 ; 
	}

	checkCudaErrors( hipMalloc((void**)&d_counter, sizeof(int)));
	checkCudaErrors( hipMalloc((void**)&d_finish, sizeof(bool)));

	checkCudaErrors( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	
	counter = 0 ;
	int *d_relabel ; 

	checkCudaErrors( hipMalloc((void**)&d_relabel,sizeof(int) ));
	
	int h_relabel = 0 ;
	
	
	int block_num = width1 / 32 ; 

	int *d_block_num; 
	
	checkCudaErrors( hipMalloc((void**)&d_block_num, sizeof(int)));
	checkCudaErrors( hipMemcpy( d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));
	
	
	int h_count_blocks = num_of_blocks ; 
	int *d_count_blocks; 
	
	checkCudaErrors( hipMalloc((void**)&d_count_blocks, sizeof(int)));
	checkCudaErrors( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
	
	h_count_blocks = 0 ;
	
	checkCudaErrors( hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));

	counter = 1 ; 
	kernel_push1_start_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
			d_sink_weight, d_push_reser,
			d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height, 
			graph_size1, width1 , height1,d_relabel, d_stochastic, d_counter, d_finish );
	int h_terminate_condition = 0 ;	
	checkCudaErrors(hipDeviceSynchronize());
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	
	//for (int i = 0 ; i < 400; i++ )
	do
	{
		if(counter%10 == 0  )
		{
			finish = true ; 
			checkCudaErrors( hipMemcpy( d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1<<<grid_push,block_push>>>(d_push_reser, s_push_reser,  d_count_blocks, d_finish, d_block_num, width1);
			checkCudaErrors( hipMemcpy( &finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
		}
		if(counter%11 == 0 )
		{
			checkCudaErrors(hipMemset(d_stochastic, 0, sizeof(int)*num_of_blocks));
			checkCudaErrors(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0 ; 
			checkCudaErrors( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_stochastic2<<<grid_push,block_push>>>(d_terminate, d_relabel_mask, d_push_reser, s_push_reser, d_stochastic, d_block_num, width1);
			
			kernel_End<<<End_grid, End_block>>>(d_terminate, d_count_blocks, d_counter);
			
			//if ( finish == false ) printf("%d \n",counter);
			if ( finish == false && counter % 121 != 0 && counter > 0 )
				h_terminate_condition++ ;

		}
		if( counter % 2 == 0 )
		{
			
			kernel_push1_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
					d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height,
					graph_size1, width1 , height1, d_stochastic, d_block_num );
			
			/*kernel_push2_stochastic<<<grid_push, block_push>>>( d_left_weight, d_right_weight, d_down_weight, d_up_weight,
						d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
						d_relabel_mask, d_graph_heightr, d_graph_heightw,
						graph_size, width, height, graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish, d_block_num) ; 
			*/
			kernel_relabel_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
					d_sink_weight, d_push_reser,/*d_pull_left, d_pull_right, d_pull_down, d_pull_up,*/
					d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height,
					graph_size1, width1 , height1, d_stochastic,d_block_num );
			
		}
		else
		{
			kernel_push1_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
					d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightw,d_graph_heightr, graph_size,width,height,
					graph_size1, width1 , height1, d_stochastic, d_block_num);
			

			/*kernel_push2_stochastic<<<grid_push, block_push>>>( d_left_weight, d_right_weight, d_down_weight, d_up_weight,
						d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
						d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height, graph_size1, 
						width1, height1, d_relabel, d_stochastic, d_counter, d_finish, d_block_num) ; 
			*/
			
			kernel_relabel_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
					d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightw,d_graph_heightr, graph_size,width,height,
					graph_size1, width1 , height1, d_stochastic, d_block_num );
			
		}
		counter++ ;
	}
	while(h_terminate_condition == 0  && counter < 500 );

	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	printf("TT Cuts :: %f %d\n",sdkGetTimerValue(&timer), counter);
	sdkDeleteTimer(&timer);
}

void bfsLabeling()
{
	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);

	checkCudaErrors( hipMemcpy( d_graph_heightr, h_graph_height, size_int, hipMemcpyHostToDevice));

	for(int i = 0 ; i < graph_size ; i++ )
		h_pixel_mask[i]=true;

	checkCudaErrors( hipMemcpy( d_pixel_mask, h_pixel_mask, sizeof(bool) * graph_size1, hipMemcpyHostToDevice));

	kernel_bfs_t<<<d_grid,d_block,0>>>(d_push_reser,d_sink_weight,d_graph_heightr,d_pixel_mask,graph_size,width,height, graph_size1, width1, height1);
	counter=1;

	checkCudaErrors( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	do
	{
		h_over=false;

		checkCudaErrors( hipMemcpy( d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) );

		kernel_bfs<<< d_grid,d_block, 0 >>>(d_left_weight,d_right_weight,d_down_weight,d_up_weight,d_graph_heightr,d_pixel_mask,
							graph_size,width,height,graph_size1, width1, height1, d_over,d_counter);

		getLastCudaError("Kernel execution failed");

		checkCudaErrors( hipMemcpy( &h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) );

		counter++;

		checkCudaErrors(hipMemcpy(d_counter,&counter,sizeof(int),hipMemcpyHostToDevice));
	}
	while(h_over);
	
	checkCudaErrors(hipMemcpy(h_graph_height,d_graph_heightr,size_int,hipMemcpyDeviceToHost));
}

int cudaCutsGetResult( )
{
	if( deviceCheck < 1 )
		return -1 ; 

	int alpha = 1 ;

	for(int i = 0 ; i < graph_size1 ; i++ ) 
	{
		int row_here = i / width1, col_here = i % width1 ;
		if(h_graph_height[i]>0 && row_here < height && row_here > 0 && col_here < width && col_here > 0 ) {
			pixelLabel[i]=alpha;
		}
	}
	
	return 0 ; 
}

int cudaCutsGetEnergy()
{
	return data_energy() + smooth_energy() ;
}

int data_energy()
{
	int eng=0;

	for(int i = 0 ; i < height ; i ++)
	{
		for(int j = 0 ; j < width ; j++)
		{
			eng += datacost(i*width+j, pixelLabel[i*width1+j]);
		}
	}

	printf("DATA ENERGY: %d\n",eng);
	return(eng);
}

int smooth_energy()
{
	int eng = 0;

	int x,y;

	for ( y = 0; y < height; y++ )
		for ( x = 1; x < width; x++ )
		{
			if( cueValues == 1 )
				eng = eng + smoothnesscost(pixelLabel[y*width1+x],pixelLabel[y*width1+x-1])*hcue[y*width+x-1];
			else
				eng = eng + smoothnesscost(pixelLabel[y*width1+x],pixelLabel[y*width1+x-1]);

		}

	for ( y = 1; y < height; y++ )
		for ( x = 0; x < width; x++ )
		{
			if(cueValues == 1)
				eng = eng + smoothnesscost(pixelLabel[y*width1+x],pixelLabel[y*width1+x-width1])*vcue[y*width+x-width];
			else
				eng = eng + smoothnesscost(pixelLabel[y*width1+x],pixelLabel[y*width1+x-width1]);

		}
	printf("SMOOTHNESS ENERGY: %d\n",eng);
	return(eng);
}


void cudaCutsFreeMem()
{
	free(h_reset_mem);
	free(h_graph_height);

	checkCudaErrors(hipFree(d_left_weight));
	checkCudaErrors(hipFree(d_right_weight));
	checkCudaErrors(hipFree(d_down_weight));
	checkCudaErrors(hipFree(d_up_weight));
	checkCudaErrors(hipFree(d_sink_weight));
	checkCudaErrors(hipFree(d_push_reser));

	checkCudaErrors(hipFree(d_pull_left));
	checkCudaErrors(hipFree(d_pull_right));
	checkCudaErrors(hipFree(d_pull_down));
	checkCudaErrors(hipFree(d_pull_up));

	checkCudaErrors(hipFree(d_graph_heightr));
	checkCudaErrors(hipFree(d_graph_heightw));
}

#endif

